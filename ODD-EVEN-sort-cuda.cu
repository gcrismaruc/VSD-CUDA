#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


long n = 21;


__global__ void testKernel(long *in, long *out, long size, long n){

	bool oddeven = true;
	__shared__ bool swappedodd;
	__shared__ bool swappedeven;
	
	long temp;

	while(1){
	
      if(n % 2 == 0) {
          
		if(oddeven == true){
			__syncthreads();

			swappedeven = false;

			__syncthreads();

			//first column only which would have the array
			if (threadIdx.y == 0) {
			
				int idx = threadIdx.x;
				
				//0, 1, 2 threads will go through
				if( idx < (size / 2) ){
					//COMPARISONS:
					// 0 <--> 1
					// 2 <--> 3
					// 4 <--> 5
					if ( in[2 * idx] > in[2 * idx + 1] ){
						//BUBBLE SORT LOGIC
						temp = in[2 * idx];
						in[2 * idx] = in[2 * idx + 1];
						in[2 * idx + 1] = temp;
						swappedeven = true;
					
					}
				}
			}
			__syncthreads();
		}
		else{

			__syncthreads();

			swappedodd=false;

			__syncthreads();

			if (threadIdx.y == 0) {
				int idx = threadIdx.x;
				//0, 1 will go through
				if( idx < (size / 2) - 1 ){
					//COMPARISONS:
					// 1 <--> 2
					// 3 <--> 4
					if ( in[2 * idx + 1] > in[2 * idx + 2] ){
						temp = in[2 * idx + 1];
						in[2 * idx + 1] = in[2 * idx + 2];
						in[2 * idx + 2] = temp;
						swappedodd = true;
					}
				}
			}
			__syncthreads();
		}
  } else {
          
          if(oddeven == true)
          {
			       __syncthreads();

			       swappedeven = false;

			       __syncthreads();

			   //first column only which would have the array
			     if (threadIdx.y == 0) 
            {
             int idx = threadIdx.x;
				     //0, 1, 2 threads will go through
				     if( idx < (size / 2) )
             {
					     if(in[2 * idx] > in[2 * idx + 1])
                 {
						        temp = in[2 * idx];
						        in[2 * idx] = in[2 * idx + 1];
						        in[2 * idx + 1] = temp;
						        swappedeven = true;
					       }
				      }
			       }
			      __syncthreads();
		      }
		  else
          {
			     __syncthreads();
			     swappedodd=false;
			     __syncthreads();

			     if (threadIdx.y == 0) 
           {
				      int idx = threadIdx.x;
				      //0, 1 will go through
				      if(idx < (size / 2))
               {
					       if(in[2 * idx + 1] > in[2 * idx + 2])
                  {
					 	          temp = in[2 * idx + 1];
						          in[2 * idx + 1] = in[2 * idx + 2];
						          in[2 * idx + 2] = temp;
						          swappedodd = true;
					         }
				      }
			     }
			     __syncthreads();
		      }
      }
	
	//if there are no swaps in odd phase as well as even phase then break (which means all sorting is done)
	// !(false) => true
	if( !( swappedodd || swappedeven ) )
		break;

	oddeven = !oddeven;	//switch phase of sorting

	}

	__syncthreads();

	//Store this phase's in[] array to out[] array
	int idx = threadIdx.x;

	if ( idx < size )
		out[idx] = in[idx];
		
}


int main(void)
{
	long i;
	long *a, *a_sorted;
	long *d_a, *d_sorted;
	//int n = 1* 1000 * 10;		//make sure to keep this even
	
    
    long size = sizeof(long) *n;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


	hipMalloc( (void**) &d_a, size);
	hipMalloc( (void**) &d_sorted, size);

	a = (long*) malloc(size);
	a_sorted = (long*) malloc(size);

	
   time_t t;
   
   
   /* Intializes random number generator */
   srand((unsigned) time(&t));
   int random_nr; 
	for(i = 0; i < n; i++) 
	{
      random_nr = rand() % 100;
		a[i] = random_nr;
           // printf(" a[%d] = %ld ", i, a[i]);

	}
	
	//d_a -> destination. a -> source.
	//Host to device array copy
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	
	//<<< >>> CUDA semantic
	testKernel<<<3 ,n>>>(d_a, d_sorted, n, n);

	//Device to Host array for final display (I/O)
	hipMemcpy(a_sorted, d_sorted, size, hipMemcpyDeviceToHost);
	
	for (i=0;i<n;i++){
		printf("%ld, ",a_sorted[i]);
	}
	
	printf("\n");
    
    hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time spent: %.5f\n", milliseconds);
    
    size_t free_byte ;

        size_t total_byte ;

        hipMemGetInfo( &free_byte, &total_byte ) ;

        
        double free_db = (double)free_byte ;

        double total_db = (double)total_byte ;

        double used_db = total_db - free_db ;

        printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
  hipEventDestroy(start);
  hipEventDestroy(stop);
	
	//free memory allocated by malloc and cudamalloc
	free(a);
	free(a_sorted);
	hipFree(d_sorted);
	hipFree(d_a);
}
